#include "hip/hip_runtime.h"
/*
FFT_GPU with shared memory
Ye Wang
*/
#include <stdlib.h>
#include <stdio.h>

#define _USE_MATH_DEFINES
#include <math.h>
//#define M_PI 3.141592653589793f

//#define N 33554432
//#define thread_num 16
//#define thread_num 32
//#define thread_num 64 
#define thread_num 128 
//#define thread_num 256 
//#define thread_num 512 
//#define N 1024
//#define N 2048
//#define N 4096
//#define N 8192
//#define N 16384
//#define N 32768
//#define N 65536
//#define N 131072
//#define N 262144
//#define N 524288
//#define N 1048576
//#define N 2097152
#define N 4194304
//#define N 8388608
//#define N 16777216
//#define N 33554432

//#define T 1024
//#define T 2048
//#define T 4096
//#define T 8192
//#define T 16384
//#define T 32768
//#define T 65536
//#define T 131072
//#define T 262144
//#define T 524288
//#define T 1048576
//#define T 2097152
//#define T 4194304
//#define T 8388608
//#define T 16777216
//#define T 33554432
float *data_real=(float*)malloc(sizeof(float)*N);
float *data_imag=(float*)malloc(sizeof(float)*N);


__global__ void fft(float* data_real_d,float* data_imag_d,int p)
{	
	
	unsigned int sub,index,elements_per_thread,i,i2,sub2,x,block,tmp2;
	float tw_real;
	float tw_imag;
	unsigned int power;
	float tmp;
	
	index=threadIdx.x+blockIdx.x*blockDim.x;  
	elements_per_thread=(N/2)/T;
	sub2=threadIdx.x*elements_per_thread;

	 __shared__ float reals[(N/2)/T*thread_num];
	 __shared__ float reals2[(N/2)/T*thread_num];
	 __shared__ float imags[(N/2)/T*thread_num];
	 __shared__ float imags2[(N/2)/T*thread_num];

		//determine which block the thread is in(not cuda block)
		power=__powf(2,p);
		x=N/(power*2);
		if(x<elements_per_thread){
            block=index*(elements_per_thread/x);
            tmp=0;
        }
		else{
            block=(index)/(x/elements_per_thread);
            tmp=(index)%(x/elements_per_thread);
		}
    
		sub=block*x*2+tmp*elements_per_thread;
		tmp2=sub;
		
		
		//issue request for all parts
	for(i=0,i2=0;i<elements_per_thread;i++)
	{	
		reals[sub2+i]=data_real_d[sub+i2];
		imags[sub2+i]=data_imag_d[sub+i2];
		reals2[sub2+i]=data_real_d[sub+x+i2];
		imags2[sub2+i]=data_imag_d[sub+x+i2];
		
		if(i2==x-1)
		{
			sub+=(x*2);
			i2=0;
		}
		else{i2++;}
		
	}
	
    sub=tmp2;

	for(i=0,i2=0;i<elements_per_thread;i++)
	{	//compute twiddle factor
		tmp=(sub+i)%(x);
		tmp=(2*M_PI*tmp*power)/N;
		tw_real=cosf(tmp);
		tw_imag=-1*sinf(tmp);
		
		//butterfly real parts
		tmp=reals[sub2+i]+reals2[sub2+i];
		reals2[sub2+i]=reals[sub2+i]-reals2[sub2+i];
		reals[sub2+i]=tmp;

		//write back real results of butterfly,only this part is written because we still need to twiddle the other
		data_real_d[sub+i2]=reals[sub2+i];
		//butterfly imag part
		tmp=imags[sub2+i]+imags2[sub2+i];
		imags2[sub2+i]=imags[sub2+i]-imags2[sub2+i];
		imags[sub2+i]=tmp;
		//multiply by twiddle
		tmp=reals2[sub2+i];
		reals2[sub2+i]=reals2[sub2+i]*tw_real-imags2[sub2+i]*tw_imag;
		data_real_d[sub+x+i2]=reals2[sub2+i];
		imags2[sub2+i]=tmp*tw_imag+imags2[sub2+i]*tw_real;
		//write back imag result of butterfly
		data_imag_d[sub+i2]=imags[sub2+i];
		data_imag_d[sub+x+i2]=imags2[sub2+i];
				
		if(i2==x-1)
		{
			sub+=x*2;
			i2=0;
		}
		else{
            i2++;
        }
    }
}



void bit_reversal()
{
   long i,i1,j,k,i2;
   double c1,c2,tx,ty;
   i2 = N >> 1;
   j = 0;
   for (i=0;i<N-1;i++) {
      if (i < j) {
         tx = data_real[i];
         ty = data_imag[i];
         data_real[i] = data_real[j];
         data_imag[i] = data_imag[j];
         data_real[j] = tx;
         data_imag[j] = ty;
      }
      k = i2;
      while (k <= j) {
         j -= k;
         k >>= 1;
      }
      j += k;
   }
}


int main( int argc, char** argv) 
{
	for(int i=0;i<N;i++)
	{	
		if(i<N/2) 
		{data_real[i]=1;
		data_imag[i]=0;}
		else{
		data_real[i]=0;
		data_imag[i]=0;
		}
	}

	int passes=(int)log2f(N);
	float* data_real_d;
	float* data_imag_d;

	hipMalloc((void**)&data_real_d,N*sizeof(float));
	hipMalloc((void**)&data_imag_d,N*sizeof(float));

	hipMemcpy(data_real_d,data_real,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(data_imag_d,data_imag,sizeof(float)*N,hipMemcpyHostToDevice);

	dim3 dimBlock(thread_num,1,1);
	dim3 dimGrid(T/thread_num,1,1);
	hipDeviceSynchronize();

	hipEvent_t start, stop; float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    hipEventRecord( start, 0 );
	for(int i=0;i<passes;i++)
	{
        fft<<<dimGrid,dimBlock>>>(data_real_d,data_imag_d,i);
    }
	hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
    printf("N=%d, T=%d, thread_num=%d\n FFT time=%f ms\n",N,T,thread_num,time);
    
	
    const char* err=hipGetErrorString(hipGetLastError());
    printf("Runtime info: %s\n",err);


	hipMemcpy(data_real,data_real_d,4*N,hipMemcpyDeviceToHost);
	hipMemcpy(data_imag,data_imag_d,4*N,hipMemcpyDeviceToHost);
	hipFree(data_real_d);
	hipFree(data_imag_d);
	bit_reversal();

for(int i=0;i<16;i++)
	{
		printf("data[%d]=%f + %f i\n",i,data_real[i],data_imag[i]);
	}

}

